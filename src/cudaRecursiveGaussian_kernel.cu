#include "hip/hip_runtime.h"
#ifndef CUDARECURSIVEGAUSSIAN_KERNEL_CU
#define CUDARECURSIVEGAUSSIAN_KERNEL_CU

#include <hip/hip_vector_types.h>
#include <stdio.h>
#include <iostream>

#define BLOCK_DIM 16

__global__ void d_transpose(int *odata, int *idata, int width, int height, int depth)
{
    __shared__ int block[BLOCK_DIM][BLOCK_DIM+1];
    //int blockIdx_y, blockIdx_x;
    // do diagonal reordering
	/*if (width == height)
	{
		blockIdx_y = blockIdx.x;
		blockIdx_x = (blockIdx.x+blockIdx.y)%gridDim.x;
	}
	else
	{
		int bid = blockIdx.x + gridDim.x*blockIdx.y;
		blockIdx_y = bid%gridDim.y;
		blockIdx_x = ((bid/gridDim.y)+blockIdx_y)%gridDim.x;
	}*/

    int xIndex, yIndex, zIndex;
    
    xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
    yIndex = (blockIdx.y * BLOCK_DIM + threadIdx.y) % height;
    zIndex = (blockIdx.y * BLOCK_DIM + threadIdx.y) / height;
    
    int slice = height * width;
    
    if((xIndex < width) && (yIndex < height) && (zIndex < depth))
    {
		block[threadIdx.y][threadIdx.x] = idata[zIndex * slice + yIndex * width + xIndex];
    }
    __syncthreads();
    
    xIndex = (blockIdx.y * BLOCK_DIM + threadIdx.y) % height;
    yIndex = (blockIdx.y * BLOCK_DIM + threadIdx.y) / height;
    zIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
    
    
    slice = height * depth;
    
    if ((xIndex < height) && (yIndex < depth) && (zIndex < width))
    {
		odata[zIndex * slice + yIndex * height + xIndex] = block[threadIdx.y][threadIdx.x];
    }
}


extern "C" void Call_d_transpose(int *odata, int *idata, int width, int height, int depth, int x, int y)
{
    dim3 grid(x, y, 1);
    dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);
    d_transpose<<< grid, threads >>>(odata, idata, width, height, depth);
}
































__global__ void d_recursiveGaussianY(int *d_src, int *d_dest, int depth, int height, int width, float b0, float b1, float b2, float b3, float B, int order, float M11, float M12, float M13, float M21, float M22, float M23, float M31, float M32, float M33)
{
	float wP1 = 0.f, wP2 = 0.f, wP3 = 0.f;
	int y = 0;
	float outF1 = 0.f, outF2 = 0.f, outF3 = 0.f;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int yy = blockIdx.y*width*height;
	if(x > width)
		return;
	
	d_src += x + yy;
	d_dest += x + yy;
	
	wP1 = (float)*d_src/sqrt(B); wP2 = wP1; wP3 = wP1;
	
	switch (order) 
	{
		case 0: 
		{
			for(y=0;y<height;y++)
			{
				float xC = (float)*d_src;
				float wC = (float)(xC - b1*wP1 - b2*wP2 - b3*wP3)/b0;
				*d_dest = (int)wC;
				d_src += width; d_dest += width;
				wP3 = wP2; wP2 = wP1; wP1 = wC;
			}
			
			d_src -= width;
			d_dest -= width;
			
			
			float up = (float)*d_src/(1.0+b1+b2+b3);
			float vp = (float)up/(1.0+b1+b2+b3);
			
			float out = 0.f;
			out = (float)M11*(*d_dest-up) + M12*(*(d_dest-width)-up) + M13*(*(d_dest-2*width)-up)+vp;
			outF1 = (float)M21*(*d_dest-up) + M22*(*(d_dest-width)-up) + M23*(*(d_dest-2*width)-up)+vp;
			outF2 = (float)M31*(*d_dest-up) + M32*(*(d_dest-width)-up) + M33*(*(d_dest-2*width)-up)+vp;
			out *= B; outF1 *= B; outF2 *= B;
			outF3 = outF2; outF2 = outF1; outF1 = out;
			
			*d_dest = (int)out;
			d_src -= width;
			d_dest -= width;
						
			for(y=height-1-1;y>=0;y--)
			{
				float wC = (float)*d_dest;
				out = (float)(B*wC - b1*outF1 - b2*outF2 - b3*outF3)/b0;
				*d_dest = (int)out;
				d_src -= width; d_dest -= width;
				outF3 = outF2; outF2 = outF1; outF1 = out;
			}
		} break;

		case 1:
		{
			float xP1 = (float)*(d_src);
			float xF1 = (float)*(d_src + 1*width);
			
			wP1 = (float)(xF1 - xP1)/2.0*(1.0+b1+b2+b3); wP3 = wP2 = wP1;
			
			float wC = ((xF1- xP1)/2.0 - b1*wP1 - b2*wP1 - b3*wP1)/b0;
			*d_dest = (int)wC;
			d_src += width; d_dest += width;
			wP3 = wP2; wP2 = wP1; wP1 = wC;
			
			for(y=0;y<height-1-1;y++)
			{
				xP1 = (float)*(d_src - width);
				xF1 = (float)*(d_src + width);
				wC = (float)((xF1- xP1)/2.0 - b1*wP1 - b2*wP2 - b3*wP3)/b0;
				*d_dest = (int)wC;
				d_src += width; d_dest += width;
				wP3 = wP2; wP2 = wP1; wP1 = wC;
			}
			
			xP1 = (float)*(d_src - width);
			xF1 = (float)*d_src;
			wC = (float)((xF1- xP1)/2.0 - b1*wP1 - b2*wP2 - b3*wP3)/b0;
			*d_dest = (int)wC;
			wP3 = wP2; wP2 = wP1; wP1 = wC;
			
			float up = (*d_src - *(d_src-width))/2.0*(1.0+b1+b2+b3);
			float vp = up/(1.0+b1+b2+b3);
			
			float out = 0.f;
			out = (float)M11*(*d_dest-up) + M12*(*(d_dest-width)-up) + M13*(*(d_dest-2*width)-up)+vp;
			outF1 = (float)M21*(*d_dest-up) + M22*(*(d_dest-width)-up) + M23*(*(d_dest-2*width)-up)+vp;
			outF2 = (float)M31*(*d_dest-up) + M32*(*(d_dest-width)-up) + M33*(*(d_dest-2*width)-up)+vp;
			out *= B; outF1 *= B; outF2 *= B;
			outF3 = outF2; outF2 = outF1; outF1 = out;
			
			*d_dest = (int)out;
			
			d_src -= width;
			d_dest -= width;
						
			for(y=height-2;y>=0;y--)
			{
				float wC = (float)*d_dest;
				out = (float)(B*wC - b1*outF1 - b2*outF2 - b3*outF3)/b0;
				*d_dest = (int)out;
				d_src -= width; d_dest -= width;
				outF3 = outF2; outF2 = outF1; outF1 = out;
			}		
		} break;

		case 2: 
		{
			float xP1 = (float)*d_src;
			float xC = (float)*d_src;
			float xF1 = (float)*(d_src+width);
			wP1 = 0.0/(1.0+b1+b2+b3); wP3 = wP2 = wP1;
			
			float wC = (float)((xF1 - 2*xC + xP1) - b1*wP1 - b2*wP2 - b3*wP3)/b0;
			*d_dest = (int)wC;
			d_src += width; d_dest += width;
			wP3 = wP2; wP2 = wP1; wP1 = wC;
			
			for(y=0;y<height-2;y++)
			{
				xC = (float)*d_src;
				xP1 = (float)*(d_src-width);
				xF1 = (float)*(d_src+width);
				wC = (float)(xF1 - 2*xC + xP1 - b1*wP1 - b2*wP2 - b3*wP3)/b0;
				*d_dest = (int)wC;
				d_src += width; d_dest += width;
				wP3 = wP2; wP2 = wP1; wP1 = wC;
			}
			
			xC = (float)*d_src;
			xP1 = (float)*(d_src-width);
			xF1 = (float)*d_src;
			wC = (float)(xF1 - 2*xC + xP1 - b1*wP1 - b2*wP2 - b3*wP3)/b0;
			*d_dest = (int)wC;
			wP3 = wP2; wP2 = wP1; wP1 = wC;
			
			float up = 0;
			float vp = up/(1.0+b1+b2+b3);
			
			float out = 0.f;
			out = (float)M11*(*d_dest-up) + M12*(*(d_dest-width)-up) + M13*(*(d_dest-2*width)-up)+vp;
			outF1 = (float)M21*(*d_dest-up) + M22*(*(d_dest-width)-up) + M23*(*(d_dest-2*width)-up)+vp;
			outF2 = (float)M31*(*d_dest-up) + M32*(*(d_dest-width)-up) + M33*(*(d_dest-2*width)-up)+vp;
			out *= B; outF1 *= B; outF2 *= B;
			outF3 = outF2; outF2 = outF1; outF1 = out;
			
			*d_dest = (int)out;
			
			d_src -= width;
			d_dest -= width;
			
			for(y=height-2;y>=0;y--)
			{
				float wC = (float)*d_dest;
				out = (float)(B*wC - b1*outF1 - b2*outF2 - b3*outF3)/b0;
				*d_dest = (int)out;
				d_src -= width; d_dest -= width;
				outF3 = outF2; outF2 = outF1; outF1 = out;
			}
		} break;
	}
}

extern "C" void Call_d_recursiveGaussianY(int *d_src, int *d_dest, int width, int height, int depth, float b0, float b1, float b2, float b3, float B, int order, int n, int nthread, float M11, float M12, float M13, float M21, float M22, float M23, float M31, float M32, float M33)
{
	d_recursiveGaussianY<<<dim3(n, depth), nthread>>>(d_src, d_dest, depth, height, width, b0, b1, b2, b3, B, order, M11, M12, M13, M21, M22, M23, M31, M32, M33);
}

__global__ void d_recursiveGaussianX(int *d_src, int *d_dest, int depth, int height, int width, float b0, float b1, float b2, float b3, float B, int order, float M11, float M12, float M13, float M21, float M22, float M23, float M31, float M32, float M33)
{
	float wP1 = 0.f, wP2 = 0.f, wP3 = 0.f;
	int y = 0;
	float outF1 = 0.f, outF2 = 0.f, outF3 = 0.f;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int yy = blockIdx.y*width*height;
	if(x > width)
		return;
		
	d_src += x*width + yy;
	d_dest += x*width + yy;

	wP1 = (float)*d_src/sqrt(B); wP2 = wP1; wP3 = wP1;
	
	switch (order) 
	{
		case 0: 
		{
			for(y=0;y<width;y++)
			{
				float xC = (float)*d_src;
				float wC = (float)(xC - b1*wP1 - b2*wP2 - b3*wP3)/b0;
				*d_dest = (int)wC;
				d_src += 1; d_dest += 1;
				wP3 = wP2; wP2 = wP1; wP1 = wC;
			}
			
			d_src -= 1;
			d_dest -= 1;
			
			
			float up = (float)*d_src/(1.0+b1+b2+b3);
			float vp = (float)up/(1.0+b1+b2+b3);
			
			float out = 0.f;
			out = (float)M11*(*d_dest-up) + M12*(*(d_dest-1)-up) + M13*(*(d_dest-2*1)-up)+vp;
			outF1 = (float)M21*(*d_dest-up) + M22*(*(d_dest-1)-up) + M23*(*(d_dest-2*1)-up)+vp;
			outF2 = (float)M31*(*d_dest-up) + M32*(*(d_dest-1)-up) + M33*(*(d_dest-2*1)-up)+vp;
			out *= B; outF1 *= B; outF2 *= B;
			outF3 = outF2; outF2 = outF1; outF1 = out;
			
			*d_dest = (int)out;
			d_src -= 1;
			d_dest -= 1;
						
			for(y=width-1-1;y>=0;y--)
			{
				float wC = (float)*d_dest;
				out = (float)(B*wC - b1*outF1 - b2*outF2 - b3*outF3)/b0;
				*d_dest = (int)out;
				d_src -= 1; d_dest -= 1;
				outF3 = outF2; outF2 = outF1; outF1 = out;
			}
		} break;

		case 1:
		{
			float xP1 = (float)*(d_src);
			float xF1 = (float)*(d_src + 1*1);
			
			wP1 = (float)(xF1 - xP1)/2.0*(1.0+b1+b2+b3); wP3 = wP2 = wP1;
			
			float wC = ((xF1- xP1)/2.0 - b1*wP1 - b2*wP1 - b3*wP1)/b0;
			*d_dest = (int)wC;
			d_src += 1; d_dest += 1;
			wP3 = wP2; wP2 = wP1; wP1 = wC;
			
			for(y=0;y<width-2;y++)
			{
				xP1 = (float)*(d_src - 1);
				xF1 = (float)*(d_src + 1);
				wC = (float)((xF1- xP1)/2.0 - b1*wP1 - b2*wP2 - b3*wP3)/b0;
				*d_dest = (int)wC;
				d_src += 1; d_dest += 1;
				wP3 = wP2; wP2 = wP1; wP1 = wC;
			}
			
			xP1 = (float)*(d_src - 1);
			xF1 = (float)*d_src;
			wC = (float)((xF1- xP1)/2.0 - b1*wP1 - b2*wP2 - b3*wP3)/b0;
			*d_dest = (int)wC;
			wP3 = wP2; wP2 = wP1; wP1 = wC;
			
			float up = (*d_src - *(d_src-1))/2.0*(1.0+b1+b2+b3);
			//float up = 0;
			float vp = up/(1.0+b1+b2+b3);
			
			float out = 0.f;
			out = (float)M11*(*d_dest-up) + M12*(*(d_dest-1)-up) + M13*(*(d_dest-2*1)-up)+vp;
			outF1 = (float)M21*(*d_dest-up) + M22*(*(d_dest-1)-up) + M23*(*(d_dest-2*1)-up)+vp;
			outF2 = (float)M31*(*d_dest-up) + M32*(*(d_dest-1)-up) + M33*(*(d_dest-2*1)-up)+vp;
			out *= B; outF1 *= B; outF2 *= B;
			outF3 = outF2; outF2 = outF1; outF1 = out;
			
			*d_dest = (int)out;
			
			d_src -= 1;
			d_dest -= 1;
						
			for(y=width-2;y>=0;y--)
			{
				float wC = (float)*d_dest;
				out = (float)(B*wC - b1*outF1 - b2*outF2 - b3*outF3)/b0;
				*d_dest = (int)out;
				d_src -= 1; d_dest -= 1;
				outF3 = outF2; outF2 = outF1; outF1 = out;
			}		
		} break;

		case 2: 
		{
			float xP1 = (float)*d_src;
			float xC = (float)*d_src;
			float xF1 = (float)*(d_src+1);
			wP1 = 0.0/(1.0+b1+b2+b3); wP3 = wP2 = wP1;
			
			float wC = (int)((xF1 - 2*xC + xP1) - b1*wP1 - b2*wP2 - b3*wP3)/b0;
			*d_dest = (int)wC;
			d_src += 1; d_dest += 1;
			wP3 = wP2; wP2 = wP1; wP1 = wC;
			
			for(y=0;y<width-2;y++)
			{
				xC = (float)*d_src;
				xP1 = (float)*(d_src-1);
				xF1 = (float)*(d_src+1);
				wC = (float)(xF1 - 2*xC + xP1 - b1*wP1 - b2*wP2 - b3*wP3)/b0;
				*d_dest = (int)wC;
				d_src += 1; d_dest += 1;
				wP3 = wP2; wP2 = wP1; wP1 = wC;
			}
			
			xC = (float)*d_src;
			xP1 = (float)*(d_src-1);
			xF1 = (float)*d_src;
			wC = (float)(xF1 - 2*xC + xP1 - b1*wP1 - b2*wP2 - b3*wP3)/b0;
			*d_dest = (int)wC;
			wP3 = wP2; wP2 = wP1; wP1 = wC;
			
			float up = 0;
			float vp = up/(1.0+b1+b2+b3);
			
			float out = 0.f;
			out = (float)M11*(*d_dest-up) + M12*(*(d_dest-1)-up) + M13*(*(d_dest-2*1)-up)+vp;
			outF1 = (float)M21*(*d_dest-up) + M22*(*(d_dest-1)-up) + M23*(*(d_dest-2*1)-up)+vp;
			outF2 = (float)M31*(*d_dest-up) + M32*(*(d_dest-1)-up) + M33*(*(d_dest-2*1)-up)+vp;
			out *= B; outF1 *= B; outF2 *= B;
			outF3 = outF2; outF2 = outF1; outF1 = out;
			
			*d_dest = (int)out;
			
			d_src -= 1;
			d_dest -= 1;
			
			for(y=width-2;y>=0;y--)
			{
				float wC = (float)*d_dest;
				out = (float)(B*wC - b1*outF1 - b2*outF2 - b3*outF3)/b0;
				*d_dest = (int)out;
				d_src -= 1; d_dest -= 1;
				outF3 = outF2; outF2 = outF1; outF1 = out;
			}
		} break;
	}
}

extern "C" void Call_d_recursiveGaussianX(int *d_src, int *d_dest, int width, int height, int depth, float b0, float b1, float b2, float b3, float B, int order, int n, int nthread, float M11, float M12, float M13, float M21, float M22, float M23, float M31, float M32, float M33)
{
	d_recursiveGaussianX<<<dim3(n, depth), nthread>>>(d_src, d_dest, depth, height, width, b0, b1, b2, b3, B, order, M11, M12, M13, M21, M22, M23, M31, M32, M33);
}

__global__ void d_recursiveGaussianZ(int* d_src, int* d_dest, int depth, int height, int width, float b0, float b1, float b2, float b3, float B, int order, float M11, float M12, float M13, float M21, float M22, float M23, float M31, float M32, float M33)
{
	float wP1 = 0.f, wP2 = 0.f, wP3 = 0.f;
	int y = 0;
	float outF1 = 0.f, outF2 = 0.f, outF3 = 0.f;
	
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int yy = blockIdx.y*width;
	if(x > width)
		return;
	
	d_src += x + yy;
	d_dest += x + yy;
	
	wP1 = (float)*d_src/sqrt(B); wP2 = wP1; wP3 = wP1;
	
	switch (order) 
	{
		case 0: 
		{
			for(y=0;y<depth;y++)
			{
				float xC = (float)*d_src;
				float wC = (float)(xC - b1*wP1 - b2*wP2 - b3*wP3)/b0;
				*d_dest = (int)wC;
				d_src += width*height; d_dest += width*height;
				wP3 = wP2; wP2 = wP1; wP1 = wC;
			}
			
			d_src -= width*height;
			d_dest -= width*height;
			
			
			float up = (float)*d_src/(1.0+b1+b2+b3);
			float vp = (float)up/(1.0+b1+b2+b3);
			
			float out = 0.f;
			out = (float)M11*(*d_dest-up) + M12*(*(d_dest-width*height)-up) + M13*(*(d_dest-2*width*height)-up)+vp;
			outF1 = (float)M21*(*d_dest-up) + M22*(*(d_dest-width*height)-up) + M23*(*(d_dest-2*width*height)-up)+vp;
			outF2 = (float)M31*(*d_dest-up) + M32*(*(d_dest-width*height)-up) + M33*(*(d_dest-2*width*height)-up)+vp;
			out *= B; outF1 *= B; outF2 *= B;
			outF3 = outF2; outF2 = outF1; outF1 = out;
			
			*d_dest = (int)out;
			d_src -= width*height;
			d_dest -= width*height;
						
			for(y=depth-1-1;y>=0;y--)
			{
				float wC = (float)*d_dest;
				out = (float)(B*wC - b1*outF1 - b2*outF2 - b3*outF3)/b0;
				*d_dest = (int)out;
				d_src -= width*height; d_dest -= width*height;
				outF3 = outF2; outF2 = outF1; outF1 = out;
			}
		} break;

		case 1:
		{
			float xP1 = (float)*(d_src);
			float xF1 = (float)*(d_src + 1*width*height);
			
			wP1 = (float)(xF1 - xP1)/2.0*(1.0+b1+b2+b3); wP3 = wP2 = wP1;
			
			float wC = ((xF1- xP1)/2.0 - b1*wP1 - b2*wP1 - b3*wP1)/b0;
			*d_dest = (int)wC;
			d_src += width*height; d_dest += width*height;
			wP3 = wP2; wP2 = wP1; wP1 = wC;
			
			for(y=0;y<depth-2;y++)
			{
				xP1 = (float)*(d_src - width*height);
				xF1 = (float)*(d_src + width*height);
				wC = (float)((xF1- xP1)/2.0 - b1*wP1 - b2*wP2 - b3*wP3)/b0;
				*d_dest = (int)wC;
				d_src += width*height; d_dest += width*height;
				wP3 = wP2; wP2 = wP1; wP1 = wC;
			}
			
			xP1 = (float)*(d_src - width*height);
			xF1 = (float)*d_src;
			wC = (float)((xF1- xP1)/2.0 - b1*wP1 - b2*wP2 - b3*wP3)/b0;
			*d_dest = (int)wC;
			wP3 = wP2; wP2 = wP1; wP1 = wC;
						
			float up = (*d_src - *(d_src-width*height))/2.0*(1.0+b1+b2+b3);
			//float up = 0;
			float vp = up/(1.0+b1+b2+b3);
			
			float out = 0.f;
			out = (float)M11*(*d_dest-up) + M12*(*(d_dest-width*height)-up) + M13*(*(d_dest-2*width*height)-up)+vp;
			outF1 = (float)M21*(*d_dest-up) + M22*(*(d_dest-width*height)-up) + M23*(*(d_dest-2*width*height)-up)+vp;
			outF2 = (float)M31*(*d_dest-up) + M32*(*(d_dest-width*height)-up) + M33*(*(d_dest-2*width*height)-up)+vp;
			out *= B; outF1 *= B; outF2 *= B;
			outF3 = outF2; outF2 = outF1; outF1 = out;
			
			*d_dest = (int)out;
			
			d_src -= width*height;
			d_dest -= width*height;
						
			for(y=depth-2;y>=0;y--)
			{
				float wC = (float)*d_dest;
				out = (float)(B*wC - b1*outF1 - b2*outF2 - b3*outF3)/b0;
				*d_dest = (int)out;
				d_src -= width*height; d_dest -= width*height;
				outF3 = outF2; outF2 = outF1; outF1 = out;
			}		
		} break;

		case 2: 
		{
			float xP1 = (float)*d_src;
			float xC = (float)*d_src;
			float xF1 = (float)*(d_src+width*height);
			wP1 = 0.0/(1.0+b1+b2+b3); wP3 = wP2 = wP1;
			
			float wC = (float)((xF1 - 2*xC + xP1) - b1*wP1 - b2*wP2 - b3*wP3)/b0;
			*d_dest = (int)wC;
			d_src += width*height; d_dest += width*height;
			wP3 = wP2; wP2 = wP1; wP1 = wC;
			
			for(y=0;y<depth-1-1;y++)
			{
				xC = (float)*d_src;
				xP1 = (float)*(d_src-width*height);
				xF1 = (float)*(d_src+width*height);
				wC = (float)(xF1 - 2*xC + xP1 - b1*wP1 - b2*wP2 - b3*wP3)/b0;
				*d_dest = (int)wC;
				d_src += width*height; d_dest += width*height;
				wP3 = wP2; wP2 = wP1; wP1 = wC;
			}
			
			xC = (float)*d_src;
			xP1 = (float)*(d_src-width*height);
			xF1 = (float)*d_src;
			wC = (float)(xF1 - 2*xC + xP1 - b1*wP1 - b2*wP2 - b3*wP3)/b0;
			*d_dest = (int)wC;
			wP3 = wP2; wP2 = wP1; wP1 = wC;
					
			float up = 0;
			float vp = up/(1.0+b1+b2+b3);
			
			float out = 0.f;
			out = (float)M11*(*d_dest-up) + M12*(*(d_dest-width*height)-up) + M13*(*(d_dest-2*width*height)-up)+vp;
			outF1 = (float)M21*(*d_dest-up) + M22*(*(d_dest-width*height)-up) + M23*(*(d_dest-2*width*height)-up)+vp;
			outF2 = (float)M31*(*d_dest-up) + M32*(*(d_dest-width*height)-up) + M33*(*(d_dest-2*width*height)-up)+vp;
			out *= B; outF1 *= B; outF2 *= B;
			outF3 = outF2; outF2 = outF1; outF1 = out;
			
			*d_dest = (int)out;
			
			d_src -= width*height;
			d_dest -= width*height;
			
			for(y=depth-2;y>=0;y--)
			{
				float wC = (float)*d_dest;
				out = (float)(B*wC - b1*outF1 - b2*outF2 - b3*outF3)/b0;
				*d_dest = (int)out;
				d_src -= width*height; d_dest -= width*height;
				outF3 = outF2; outF2 = outF1; outF1 = out;
			}
		} break;
	}
}

extern "C" void Call_d_recursiveGaussianZ(int *d_src, int *d_dest, int width, int height, int depth, float b0, float b1, float b2, float b3, float B, int order, int n, int nthread, float M11, float M12, float M13, float M21, float M22, float M23, float M31, float M32, float M33)
{
	d_recursiveGaussianZ<<<dim3(n, height), nthread>>>(d_src, d_dest, depth, height, width, b0, b1, b2, b3, B, order, M11, M12, M13, M21, M22, M23, M31, M32, M33);
}


#endif //CUDARECURSIVEGAUSSIAN_KERNEL_CU

