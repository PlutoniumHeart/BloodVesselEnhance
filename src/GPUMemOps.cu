#include "GPUMemOps.cuh"

CGPUMemOps::CGPUMemOps()
{
	d_image = NULL;
	d_destination = NULL;
}

CGPUMemOps::~CGPUMemOps()
{
}

void CGPUMemOps::PassToDevice(int* h_data, int size)
{
	size_t memSize = size * sizeof(int);
	checkCudaErrors(hipMalloc((void**)&d_image, memSize));
	checkCudaErrors(hipMalloc((void**)&d_destination, memSize));
	checkCudaErrors(hipMemcpy(d_image, h_data, memSize, hipMemcpyHostToDevice));
}

void CGPUMemOps::PassToHost(int* h_data, int size, int s)
{
	size_t memSize = size * sizeof(int);
	if(s == 0)
	{
		checkCudaErrors(hipMemcpy(h_data, d_destination, memSize, hipMemcpyDeviceToHost));
	}
	else
	{
		checkCudaErrors(hipMemcpy(h_data, d_image, memSize, hipMemcpyDeviceToHost));
	}
}

void CGPUMemOps::PassToDevice(short* h_data, int size)
{
	size_t memSize = size * sizeof(short);
	checkCudaErrors(hipMalloc((void**)&d_image, memSize));
	checkCudaErrors(hipMalloc((void**)&d_destination, memSize));
	checkCudaErrors(hipMemcpy(d_image, h_data, memSize, hipMemcpyHostToDevice));
}

void CGPUMemOps::PassToHost(short* h_data, int size, int s)
{
	size_t memSize = size * sizeof(short);
	if(s == 0)
	{
		checkCudaErrors(hipMemcpy(h_data, d_destination, memSize, hipMemcpyDeviceToHost));
	}
	else
	{
		checkCudaErrors(hipMemcpy(h_data, d_image, memSize, hipMemcpyDeviceToHost));
	}
}

//void CGPUMemOps::PassToTexture(int* h_data, int width, int height, int depth, hipChannelFormatDesc channelDesc)
//{
//	size_t memSize = width * height * depth * sizeof(int);
//	hipExtent volumeSize;
//	volumeSize.width = width;
//	volumeSize.height = height;
//	volumeSize.depth = depth;
//	hipMalloc3DArray(&cuArray, &channelDesc, width, volumeSize);
//	hipMemcpy3D();
//	hipMemcpyToArray(cuArray, 0, 0, h_data, memSize, hipMemcpyHostToDevice);
//}

int* CGPUMemOps::GetInputImage()
{
	return d_image;
}

int* CGPUMemOps::GetOutputImage()
{
	return d_destination;
}

short* CGPUMemOps::GetShortInput()
{
	return d_shortImage;
}

short* CGPUMemOps::GetShortOutput()
{
	return d_shortDest;
}

void CGPUMemOps::ClearDevice()
{
	hipFree(d_destination);
	hipFree(d_image);
}